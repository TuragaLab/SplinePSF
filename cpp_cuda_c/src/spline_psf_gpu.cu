#include "hip/hip_runtime.h"
//
//  Created by Lucas Müller on 12.02.2020
//  Copyright © 2020 Lucas-Raphael Müller. All rights reserved.
//
#include <assert.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <stdexcept>
#include <sstream>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "spline_psf_gpu.cuh"
using namespace spline_psf_gpu;


// internal declarations
void check_host_coeff(const float *h_coeff);

auto forward_rois(spline *d_sp, float *d_rois, const int n, const int roi_size_x, const int roi_size_y,
    const float *d_x, const float *d_y, const float *d_z, const float *d_phot) -> void;

auto forward_drv_rois(spline *d_sp, float *d_rois, float *d_drv_rois, const int n, const int roi_size_x, const int roi_size_y,
    const float *d_x, const float *d_y, const float *d_z, const float *d_phot, const float *d_bg, const bool add_bg) -> void;

__device__
auto kernel_computeDelta3D(spline *sp,
    float* delta_f, float* delta_dxf, float* delta_dyf, float* delta_dzf,
    float x_delta, float y_delta, float z_delta) -> void;

__global__
auto kernel_derivative(spline *sp, float *rois, float *drv_rois, const int roi_ix, const int npx,
    const int npy, int xc, int yc, int zc, const float phot, const float bg,
    const float x_delta, const float y_delta, const float z_delta, const bool add_bg) -> void;

__global__
auto fAt3Dj(spline *sp, float* rois, int roi_ix, int npx, int npy,
    int xc, int yc, int zc, float phot, float x_delta, float y_delta, float z_delta) -> void;

__global__
auto kernel_roi(spline *sp, float *rois, const int npx, const int npy,
    const float* xc_, const float* yc_, const float* zc_, const float* phot_) -> void;

__global__
auto kernel_derivative_roi(spline *sp, float *rois, float *drv_rois, const int npx, const int npy,
    const float *xc_, const float *yc_, const float *zc_,
    const float *phot_, const float *bg_, const bool add_bg) -> void;

__global__
auto roi_accumulate(float *frames, const int frame_size_x, const int frame_size_y, const int n_frames,
    const float *rois, const int n_rois,
    const int *frame_ix, const int *x0, const int *y0,
    const int roi_size_x, const int roi_size_y) -> void;

namespace spline_psf_gpu {

    // check cuda availability by device count
    auto cuda_is_available(void) -> bool {

        int d_count = 0;
        hipError_t err = hipGetDeviceCount(&d_count);

        if (err != hipSuccess) {
            return false;
        }

        float min_compute_cap = 3.7;

        bool at_least_one_device = false;
        for (int i = 0; i < d_count; i++) {
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, i);

            float compute_cap = prop.major + prop.minor / 10;

            if (compute_cap >= min_compute_cap) {
                at_least_one_device = true;
                break;
            }

        }

        if (at_least_one_device) {
            return true;
        }

        return false;
    }

    // Create struct and ship it to device
    auto d_spline_init(const float *h_coeff, int xsize, int ysize, int zsize, int device_ix) -> spline* {

        // allocate struct on host and ship it to device later
        // ToDo: C++11ify this
        spline* sp;
        sp = (spline *)malloc(sizeof(spline));

        sp->xsize = xsize;
        sp->ysize = ysize;
        sp->zsize = zsize;

        sp->roi_out_eps = 1e-10;
        sp->roi_out_deriv_eps = 0.0;

        sp->n_par = 5;
        sp->n_coeff = 64;

        int tsize = xsize * ysize * zsize * 64;

        hipSetDevice(device_ix);

        float *d_coeff;
        hipMalloc(&d_coeff, tsize * sizeof(float));
        hipMemcpy(d_coeff, h_coeff, tsize * sizeof(float), hipMemcpyHostToDevice);

        sp->coeff = d_coeff;  // for some reason this should happen here and not d_sp->coeff = d_coeff ...

        // ship to device
        spline* d_sp;
        hipMalloc(&d_sp, sizeof(spline));
        hipMemcpy(d_sp, sp, sizeof(spline), hipMemcpyHostToDevice);

        // delete on host
        free(sp);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during allocation of spline struct on device.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }

        return d_sp;
    }

    auto destructor(spline *d_sp) -> void {

        // first create host helper to be able to access the pointer to coeff, dereferencing d_sp is illegal
        spline *sp;
        sp = (spline*)malloc(sizeof(spline));
        hipMemcpy(sp, d_sp, sizeof(spline), hipMemcpyDeviceToHost);

        hipFree(sp->coeff);
        free(sp);
        hipFree(d_sp);

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during destruxtor.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }
    }


    // Wrapper function to compute the ROIs on the device.
    // Takes in all the host arguments and returns leaves the ROIs on the device
    //
    auto forward_rois_host2device(spline *d_sp, const int n, const int roi_size_x, const int roi_size_y,
    const float *h_x, const float *h_y, const float *h_z, const float *h_phot) -> float* {

        hipError_t err;

        // allocate and copy coordinates and photons
        float *d_x, *d_y, *d_z, *d_phot;
        hipMalloc(&d_x, n * sizeof(float));
        hipMalloc(&d_y, n * sizeof(float));
        hipMalloc(&d_z, n * sizeof(float));
        hipMalloc(&d_phot, n * sizeof(float));
        hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_z, h_z, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_phot, h_phot, n * sizeof(float), hipMemcpyHostToDevice);

        // allocate space for rois on device
        float* d_rois;
        hipMalloc(&d_rois, n * roi_size_x * roi_size_y * sizeof(float));

        err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during ROI memory allocation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }

        hipMemset(d_rois, 0.0, n * roi_size_x * roi_size_y * sizeof(float));

        #if DEBUG
            check_spline<<<1,1>>>(d_sp);
            hipDeviceSynchronize();
        #endif

        // call to actual implementation
        forward_rois(d_sp, d_rois, n, roi_size_x, roi_size_y, d_x, d_y, d_z, d_phot);

        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_z);
        hipFree(d_phot);

        return d_rois;
    }

    // Wrapper function to ocmpute the ROIs on the device and ships it back to the host
    // Takes in all the host arguments and returns the ROIs to the host
    // Allocation for rois must have happened outside
    //
    auto forward_rois_host2host(spline *d_sp, float *h_rois, const int n, const int roi_size_x, const int roi_size_y,
        const float *h_x, const float *h_y, const float *h_z, const float *h_phot) -> void {

        auto d_rois = forward_rois_host2device(d_sp, n, roi_size_x, roi_size_y, h_x, h_y, h_z, h_phot);

        hipMemcpy(h_rois, d_rois, n * roi_size_x * roi_size_y * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_rois);
        return;
    }

    auto forward_drv_rois_host2device(spline *d_sp, float *d_rois, float *d_drv_rois, const int n, const int roi_size_x, const int roi_size_y,
        const float *h_x, const float *h_y, const float *h_z, const float *h_phot, const float *h_bg, const bool add_bg) -> void {

        // allocate and copy coordinates and photons
        float *d_x, *d_y, *d_z, *d_phot, *d_bg;
        hipMalloc(&d_x, n * sizeof(float));
        hipMalloc(&d_y, n * sizeof(float));
        hipMalloc(&d_z, n * sizeof(float));
        hipMalloc(&d_phot, n * sizeof(float));
        hipMalloc(&d_bg, n * sizeof(float));

        hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_z, h_z, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_phot, h_phot, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_bg, h_bg, n * sizeof(float), hipMemcpyHostToDevice);

        const int n_par = 5;
        hipMemset(d_rois, 0.0, n * roi_size_x * roi_size_y * sizeof(float));
        hipMemset(d_drv_rois, 0.0, n_par * n * roi_size_x * roi_size_y * sizeof(float));

        // call to actual implementation
        forward_drv_rois(d_sp, d_rois, d_drv_rois, n, roi_size_x, roi_size_y, d_x, d_y, d_z, d_phot, d_bg, add_bg);

        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_z);
        hipFree(d_phot);
        hipFree(d_bg);

        return;
    }

    auto forward_drv_rois_host2host(spline *d_sp, float *h_rois, float *h_drv_rois, const int n, const int roi_size_x, const int roi_size_y,
        const float *h_x, const float *h_y, const float *h_z, const float *h_phot, const float *h_bg, const bool add_bg) -> void {

        hipError_t err;

        // allocate space for rois and derivatives on device
        const int n_par = 5;
        float *d_rois, *d_drv_rois;

        hipMalloc(&d_rois, n * roi_size_x * roi_size_y * sizeof(float));
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during ROI memory allocation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }

        hipMalloc(&d_drv_rois, n_par * n * roi_size_x * roi_size_y * sizeof(float));
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during derivative ROI memory allocation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }

        // forward
        forward_drv_rois_host2device(d_sp, d_rois, d_drv_rois, n, roi_size_x, roi_size_y, h_x, h_y, h_z, h_phot, h_bg, add_bg);

        hipMemcpy(h_rois, d_rois, n * roi_size_x * roi_size_y * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_drv_rois, d_drv_rois, n * n_par * roi_size_x * roi_size_y * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_rois);
        hipFree(d_drv_rois);

        return;
    }

    auto forward_frames_host2host(spline *d_sp, float *h_frames, const int frame_size_x, const int frame_size_y, const int n_frames,
        const int n_rois, const int roi_size_x, const int roi_size_y,
        const int *h_frame_ix, const float *h_xr0, const float *h_yr0, const float *h_z0,
        const int *h_x_ix, const int *h_y_ix, const float *h_phot) -> void {

        auto d_frames = forward_frames_host2device(d_sp, frame_size_x, frame_size_y, n_frames,
            n_rois, roi_size_x, roi_size_y, h_frame_ix, h_xr0, h_yr0, h_z0, h_x_ix, h_y_ix, h_phot);

        hipMemcpy(h_frames, d_frames, n_frames * frame_size_x * frame_size_y * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_frames);
        return;
    }

    auto forward_frames_host2device(spline *d_sp, const int frame_size_x, const int frame_size_y, const int n_frames,
        const int n_rois, const int roi_size_x, const int roi_size_y,
        const int *h_frame_ix, const float *h_xr0, const float *h_yr0, const float *h_z0,
        const int *h_x_ix, const int *h_y_ix, const float *h_phot) -> float* {

        hipError_t err;

        // ToDo: maybe convert to stream
        float* d_frames;
        hipMalloc(&d_frames, n_frames * frame_size_x * frame_size_y * sizeof(float));
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during Frame memory allocation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }
        hipMemset(d_frames, 0.0, n_frames * frame_size_x * frame_size_y * sizeof(float));

        // allocate indices
        int *d_xix, *d_yix, *d_fix;
        hipMalloc(&d_xix, n_rois * sizeof(int));
        hipMalloc(&d_yix, n_rois * sizeof(int));
        hipMalloc(&d_fix, n_rois * sizeof(int));
        hipMemcpy(d_xix, h_x_ix, n_rois * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_yix, h_y_ix, n_rois * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_fix, h_frame_ix, n_rois * sizeof(int), hipMemcpyHostToDevice);

        auto d_rois = forward_rois_host2device(d_sp, n_rois, roi_size_x, roi_size_y, h_xr0, h_yr0, h_z0, h_phot);

        // accumulate rois into frames
        const int blocks = (n_rois * roi_size_x * roi_size_y) / 256 + 1;
        const int thread_p_block = 256;
        roi_accumulate<<<blocks, thread_p_block>>>(d_frames, frame_size_x, frame_size_y, n_frames,
            d_rois, n_rois, d_fix, d_xix, d_yix, roi_size_x, roi_size_y);

        hipDeviceSynchronize();

        err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream rt_err;
            rt_err << "Error during frame computation computation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
            throw std::runtime_error(rt_err.str());
        }

        hipFree(d_xix);
        hipFree(d_yix);
        hipFree(d_fix);
        hipFree(d_rois);

        return d_frames;
    }
} // namespace spline_psf_gpu


auto forward_rois(spline *d_sp, float *d_rois, const int n, const int roi_size_x, const int roi_size_y,
    const float *d_x, const float *d_y, const float *d_z, const float *d_phot) -> void {

    // init cuda_err
    hipError_t err = hipSuccess;

    // start n blocks which itself start threads corresponding to the number of px childs (dynamic parallelism)
    kernel_roi<<<n, 1>>>(d_sp, d_rois, roi_size_x, roi_size_y, d_x, d_y, d_z, d_phot);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::stringstream rt_err;
        rt_err << "Error during ROI computation computation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
        throw std::runtime_error(rt_err.str());
    }

    return;
}

auto forward_drv_rois(spline *d_sp, float *d_rois, float *d_drv_rois, const int n, const int roi_size_x, const int roi_size_y,
    const float *d_x, const float *d_y, const float *d_z, const float *d_phot, const float *d_bg, const bool add_bg) -> void {

    // init cuda_err
    hipError_t err = hipSuccess;

    // start n blocks which itself start threads corresponding to the number of px childs (dynamic parallelism)
    kernel_derivative_roi<<<n, 1>>>(d_sp, d_rois, d_drv_rois, roi_size_x, roi_size_y, d_x, d_y, d_z, d_phot, d_bg, add_bg);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err != hipSuccess) {
        std::stringstream rt_err;
        rt_err << "Error during ROI derivative computation computation.\nCode: "<< err << "\nInformation: \n" << hipGetErrorString(err);
        throw std::runtime_error(rt_err.str());
    }

    return;
}

// Just a dummy for checking correct parsing from python
// ... had to learn the hard way ...
__global__
auto check_spline(spline *d_sp) -> void {
    printf("Checking spline ...\n");
    printf("\txs, ys, zs: %i %i %i\n", d_sp->xsize, d_sp->ysize, d_sp->zsize);
    printf("\toutside-roi value: %f\n", d_sp->roi_out_eps);
    printf("\toutside-roi derivative value: %f\n", d_sp->roi_out_deriv_eps);

    printf("\tDevice coeff: \n");
    for (int i = 0; i < 100; i++) {
        printf("\t\ti: %d coeff %f\n", d_sp->coeff[i]);
    }
    printf("\n");
}

// kernel to compute common term for spline function (for all pixels this will stay the same)
__device__
auto kernel_computeDelta3D(spline *sp, float* delta_f, float* delta_dxf, float* delta_dyf, float* delta_dzf,
    float x_delta, float y_delta, float z_delta) -> void {

    int i,j,k;
    float cx,cy,cz;

    cz = 1.0;
    for(i=0;i<4;i++){
        cy = 1.0;
        for(j=0;j<4;j++){
            cx = 1.0;
            for(k=0;k<4;k++){
                delta_f[i*16+j*4+k] = cz * cy * cx;
                if(k<3){
					delta_dxf[i*16+j*4+k+1] = ((float)k+1) * cz * cy * cx;
				}
				if(j<3){
					delta_dyf[i*16+(j+1)*4+k] = ((float)j+1) * cz * cy * cx;
				}
				if(i<3){
					delta_dzf[(i+1)*16+j*4+k] = ((float)i+1) * cz * cy * cx;
				}
                cx = cx * x_delta;
            }
            cy = cy * y_delta;
        }
        cz= cz * z_delta;
    }
}

// kernel to compute pixel-wise term
__global__
auto fAt3Dj(spline *sp, float* rois, const int roi_ix, const int npx, const int npy,
    int xc, int yc, int zc, float phot, float x_delta, float y_delta, float z_delta) -> void {

    const int i = (blockIdx.x * blockDim.x + threadIdx.x) / npx;
    const int j = (blockIdx.x * blockDim.x + threadIdx.x) % npx;

     // allocate space for df, dxf, dyf, dzf
    __shared__ float delta_f[64];
    __shared__ float dxf[64];
    __shared__ float dyf[64];
    __shared__ float dzf[64];

    // term common to all pixels, must be executed at least once per kernel block (since sync only syncs within block)
    // if (i == 0 and j == 0) {  // linear / C++ equivalent
    if (threadIdx.x == 0) {

        for (int k = 0; k < 64; k++) {
            delta_f[k] = 0.0;
            dxf[k] = 0.0;
            dyf[k] = 0.0;
            dzf[k] = 0.0;
        }

        // This is different to the C library since we needed to rearrange a bit to account for the GPU parallelism
        kernel_computeDelta3D(sp, delta_f, dxf, dyf, dzf, x_delta, y_delta, z_delta);
    }

    __syncthreads();  // wait so that all threads see the deltas. REMINDER: only works for within block

    // kill excess threads (I think it needs to happen after syncthreads)
    if ((i >= npx) || (j >= npy)) {
        return;
    }

    xc = xc + i;
    yc = yc + j;

    // If the lateral position is outside the calibration, return epsilon value
    if ((xc < 0) || (xc > sp->xsize-1) || (yc < 0) || (yc > sp->ysize-1)) {

        rois[roi_ix * npx * npy + i * npy + j] = sp->roi_out_eps;
        return;
    }

    zc = max(zc,0);
    zc = min(zc,sp->zsize-1);

    float fv = 0.0;

    for (int k = 0; k < 64; k++) {
        fv += delta_f[k] * sp->coeff[k * (sp->xsize * sp->ysize * sp->zsize) + zc * (sp->xsize * sp->ysize) + yc * sp->xsize + xc];
    }

    // write to global roi stack
    rois[roi_ix * npx * npy + i * npy + j] = phot * fv;
    return;
}

// kernel to compute psf for a single emitter
__global__
auto kernel_roi(spline *sp, float *rois, const int npx, const int npy, const float* xc_, const float* yc_, const float* zc_, const float* phot_) -> void {

    int r = blockIdx.x;  // roi number 'r'

    int x0, y0, z0;
    float x_delta,y_delta,z_delta;

    float xc = xc_[r];
    float yc = yc_[r];
    float zc = zc_[r];
    float phot = phot_[r];

    /* Compute delta. Will be the same for all following px */
    x0 = (int)floorf(xc);
    x_delta = xc - x0;

    y0 = (int)floorf(yc);
    y_delta = yc - y0;

    z0 = (int)floorf(zc);
    z_delta = zc - z0;

    int n_threads = min(1024, npx * npy);  // max number of threads per block
    int n_blocks = ceilf(static_cast<float>(npx * npy) / static_cast<float>(n_threads));

    fAt3Dj<<<n_blocks, n_threads>>>(sp, rois, r, npx, npy, x0, y0, z0, phot, x_delta, y_delta, z_delta);

    return;
}

__global__
auto kernel_derivative_roi(spline *sp, float *rois, float *drv_rois, const int npx, const int npy,
    const float *xc_, const float *yc_, const float *zc_, const float *phot_, const float *bg_, const bool add_bg) -> void {

    int r = blockIdx.x;  // roi number 'r'

    int x0, y0, z0;
    float x_delta,y_delta,z_delta;

    float xc = xc_[r];
    float yc = yc_[r];
    float zc = zc_[r];
    float phot = phot_[r];
    float bg = bg_[r];

    /* Compute delta. Will be the same for all following px */
    x0 = (int)floorf(xc);
    x_delta = xc - x0;

    y0 = (int)floorf(yc);
    y_delta = yc - y0;

    z0 = (int)floorf(zc);
    z_delta = zc - z0;

    int n_threads = min(1024, npx * npy);  // max number of threads per block
    int n_blocks = ceilf(static_cast<float>(npx * npy) / static_cast<float>(n_threads));

    kernel_derivative<<<n_blocks, n_threads>>>(sp, rois, drv_rois, r, npx, npy, x0, y0, z0, phot, bg, x_delta, y_delta, z_delta, add_bg);

    return;
}

__global__
auto kernel_derivative(spline *sp, float *rois, float *drv_rois, const int roi_ix, const int npx, const int npy,
    int xc, int yc, int zc, const float phot, const float bg, const float x_delta, const float y_delta, const float z_delta, const bool add_bg) -> void {

    int i = (blockIdx.x * blockDim.x + threadIdx.x) / npx;
    int j = (blockIdx.x * blockDim.x + threadIdx.x) % npx;

     // allocate space for df, dxf, dyf, dzf
    __shared__ float delta_f[64];
    __shared__ float dxf[64];
    __shared__ float dyf[64];
    __shared__ float dzf[64];

    float dudt[5] = { 0 };  // derivatives in this very pixel

    // term common to all pixels, must be executed at least once per kernel block (since sync only syncs within block)
    // if (i == 0 and j == 0) {  // linear / C++ equivalent
    if (threadIdx.x == 0) {

        for (int k = 0; k < 64; k++) {
            delta_f[k] = 0.0;
            dxf[k] = 0.0;
            dyf[k] = 0.0;
            dzf[k] = 0.0;
        }

        // This is different to the C library since we needed to rearrange a bit to account for the GPU parallelism
        kernel_computeDelta3D(sp, delta_f, dxf, dyf, dzf, x_delta, y_delta, z_delta);
    }
    __syncthreads();  // wait so that all threads see the deltas

    // kill excess threads
    if ((i >= npx) || (j >= npy)) {
        return;
    }

    // let each thread go to their respective pixel
    xc = xc + i;
    yc = yc + j;

    // set epsilon values outside of ROI
    if ((xc < 0) || (xc > sp->xsize-1) || (yc < 0) || (yc > sp->ysize-1)) {

        for (int k = 0; k < sp->n_par; k++) {
            dudt[k] = sp->roi_out_deriv_eps;
        }

        if (add_bg) {
            rois[roi_ix * npx * npy + i * npy + j] = sp->roi_out_eps + bg;
        }
        else {
            rois[roi_ix * npx * npy + i * npy + j] = sp->roi_out_eps;
        }
        return;
    }

    // safety for zc
    zc = max(zc,0);
    zc = min(zc,sp->zsize-1);

    float fv;  // taken from yiming, not entirely understood by myself

    // actual derivative computation
    for (int k = 0; k < 64; k++)
    {
        fv += delta_f[k] * sp->coeff[k * (sp->xsize * sp->ysize * sp->zsize) + zc * (sp->xsize * sp->ysize) + yc * sp->xsize + xc];
        dudt[0] += dxf[k] * sp->coeff[k * (sp->xsize * sp->ysize * sp->zsize) + zc * (sp->xsize * sp->ysize) + yc * sp->xsize + xc];
        dudt[1] += dyf[k] * sp->coeff[k * (sp->xsize * sp->ysize * sp->zsize) + zc * (sp->xsize * sp->ysize) + yc * sp->xsize + xc];
        dudt[4] += dzf[k] * sp->coeff[k * (sp->xsize * sp->ysize * sp->zsize) + zc * (sp->xsize * sp->ysize) + yc * sp->xsize + xc];
    }

    dudt[0] *= -1 * phot;
    dudt[1] *= -1 * phot;
    dudt[4] *= phot;
    dudt[2] = fv;
    dudt[3] = 1;

    // write to global roi and derivate stack
    if (add_bg) {
        rois[roi_ix * npx * npy + i * npy + j] = phot * fv + bg;
    }
    else {
        rois[roi_ix * npx * npy + i * npy + j] = phot * fv;
    }

    for (int k = 0; k < sp->n_par; k++) {
        drv_rois[roi_ix * sp->n_par * npx * npy + k * npx * npy + i * npy + j] = dudt[k];
    }

    return;
}

// accumulate rois to frames
__global__
auto roi_accumulate(float *frames, const int frame_size_x, const int frame_size_y, const int n_frames,
                    const float *rois, const int n_rois,
                    const int *frame_ix, const int *x0, const int *y0,
                    const int roi_size_x, const int roi_size_y) -> void {

        // kernel ix
        const long kx = (blockIdx.x * blockDim.x + threadIdx.x);
        if (kx >= n_rois * roi_size_x * roi_size_y) {
            return;
        }

        // roi index
        const long j = kx % roi_size_y;
        const long i = ((kx - j) / roi_size_y) % roi_size_x;
        const long r = (((kx - j) / roi_size_y) - i) / roi_size_x;

        const long ii = x0[r] + i;
        const long jj = y0[r] + j;

        if ((frame_ix[r] < 0) || (frame_ix[r] >= n_frames)) {  // if frame ix is outside
            return;
        }

        if ((ii < 0) || (jj < 0) || (ii >= frame_size_x) || (jj >= frame_size_y)) {  // if outside frame throw away
            return;
        }
        float val = rois[r * roi_size_x * roi_size_y + i * roi_size_y + j];
        atomicAdd(&frames[frame_ix[r] * frame_size_x * frame_size_y + ii * frame_size_y + jj], val);  // otherwise race condition

        return;
    }


